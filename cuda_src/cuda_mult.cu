#include "hip/hip_runtime.h"
#ifndef CUDA_MULT_H
#define CUDA_MULT_H

#include <stdio.h>
#include <time.h>
#include <math.h>

#include "cpoly.h"

__global__ void kernel_triv_mult(float * A, float * B, float * C, int n, int m)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
  
  if (i >= n || j >= m) return;

  float val = A[i] * B[j];
  atomicAdd(&(C[i + j]), (float)val);

}

float para_triv_mult(CPoly * A, CPoly * B, CPoly * C)
{
	int grid_x = (int)ceil((float)A->m_len / 16.0);
	int grid_y = (int)ceil((float)B->m_len / 32.0);

	dim3 dim_block(16, 32);
	dim3 dim_grid(grid_x, grid_y);

	hipEvent_t start;
	hipEvent_t stop;
	float elapsed_time = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	float * dev_a;
	float * dev_b;
	float * dev_c;

	hipMalloc((void**)&dev_a, A->m_len * sizeof(float));
	hipMalloc((void**)&dev_b, B->m_len * sizeof(float));
	hipMalloc((void**)&dev_c, C->m_len * sizeof(float));

	hipMemcpy(dev_a, (float *)A->m_coefs, A->m_len * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_b, (float *)B->m_coefs, B->m_len * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_c, (float *)C->m_coefs, C->m_len * sizeof(float),
		hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	kernel_triv_mult<<<dim_grid, dim_block>>>(dev_a, dev_b, dev_c, A->m_len, B->m_len);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);

	hipMemcpy(C->m_coefs, dev_c, C->m_len * sizeof(float), hipMemcpyDeviceToHost);

	return elapsed_time;
}

#endif